//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <viskores/cont/cuda/internal/DeviceAdapterTimerImplementationCuda.h>

#include <viskores/Types.h>
#include <viskores/cont/cuda/ErrorCuda.h>

#include <hip/hip_runtime.h>

namespace viskores
{
namespace cont
{

DeviceAdapterTimerImplementation<
  viskores::cont::DeviceAdapterTagCuda>::DeviceAdapterTimerImplementation()
{
  VISKORES_CUDA_CALL(hipEventCreate(&this->StartEvent));
  VISKORES_CUDA_CALL(hipEventCreate(&this->StopEvent));
  this->Reset();
}

DeviceAdapterTimerImplementation<
  viskores::cont::DeviceAdapterTagCuda>::~DeviceAdapterTimerImplementation()
{
  // These aren't wrapped in VISKORES_CUDA_CALL because we can't throw errors
  // from destructors. We're relying on hipGetLastError in the
  // VISKORES_CUDA_CHECK_ASYNCHRONOUS_ERROR catching any issues from these calls
  // later.
  hipEventDestroy(this->StartEvent);
  hipEventDestroy(this->StopEvent);
}

void DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Reset()
{
  this->StartReady = false;
  this->StopReady = false;
}

void DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Start()
{
  VISKORES_CUDA_CALL(hipEventRecord(this->StartEvent, hipStreamPerThread));
  this->StartReady = true;
}

void DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Stop()
{
  VISKORES_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
  VISKORES_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  this->StopReady = true;
}

bool DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Started() const
{
  return this->StartReady;
}

bool DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Stopped() const
{
  return this->StopReady;
}

// Callbacks without a mandated order(in independent streams) execute in undefined
// order and maybe serialized. So Instead CudaEventQuery is used here.
// Ref link: https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__STREAM.html
bool DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::Ready() const
{
  if (hipEventQuery(this->StopEvent) == hipSuccess)
  {
    return true;
  }
  return false;
}


viskores::Float64
DeviceAdapterTimerImplementation<viskores::cont::DeviceAdapterTagCuda>::GetElapsedTime() const
{
  assert(this->StartReady);
  if (!this->StartReady)
  {
    VISKORES_LOG_F(viskores::cont::LogLevel::Error,
                   "Start() function should be called first then trying to call GetElapsedTime().");
    return 0;
  }
  if (!this->StopReady)
  {
    // Stop was not called, so we have to insert a new event into the stream
    VISKORES_CUDA_CALL(hipEventRecord(this->StopEvent, hipStreamPerThread));
    VISKORES_CUDA_CALL(hipEventSynchronize(this->StopEvent));
  }

  float elapsedTimeMilliseconds;
  VISKORES_CUDA_CALL(
    hipEventElapsedTime(&elapsedTimeMilliseconds, this->StartEvent, this->StopEvent));
  // Reset Stop flag to its original state
  return static_cast<viskores::Float64>(0.001f * elapsedTimeMilliseconds);
}
}
} // namespace viskores::cont
