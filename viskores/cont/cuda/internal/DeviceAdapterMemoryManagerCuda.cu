//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <viskores/cont/cuda/DeviceAdapterCuda.h>
#include <viskores/cont/cuda/ErrorCuda.h>
#include <viskores/cont/cuda/internal/CudaAllocator.h>
#include <viskores/cont/cuda/internal/DeviceAdapterMemoryManagerCuda.h>

#include <viskores/cont/ErrorBadAllocation.h>

#include <viskores/Math.h>

namespace
{

void* CudaAllocate(viskores::BufferSizeType size)
{
  try
  {
    return viskores::cont::cuda::internal::CudaAllocator::Allocate(static_cast<std::size_t>(size));
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << size << " bytes on CUDA device: " << error.what();
    throw viskores::cont::ErrorBadAllocation(err.str());
  }
}

void CudaDelete(void* memory)
{
  if (memory != nullptr)
  {
    viskores::cont::cuda::internal::CudaAllocator::Free(memory);
  }
};

void CudaReallocate(void*& memory,
                    void*& container,
                    viskores::BufferSizeType oldSize,
                    viskores::BufferSizeType newSize)
{
  VISKORES_ASSERT(memory == container);

  if (newSize > oldSize)
  {
    // Make a new buffer
    void* newMemory = CudaAllocate(newSize);

    // Copy the data to the new buffer
    VISKORES_CUDA_CALL(hipMemcpyAsync(newMemory,
                                       memory,
                                       static_cast<std::size_t>(oldSize),
                                       hipMemcpyDeviceToDevice,
                                       hipStreamPerThread));

    // Reset the buffer in the passed in info
    memory = container = newMemory;
  }
  else
  {
    // Just reuse the buffer.
  }
}

} // anonymous namespace

namespace viskores
{
namespace cont
{
namespace internal
{

viskores::cont::internal::BufferInfo DeviceAdapterMemoryManager<
  viskores::cont::DeviceAdapterTagCuda>::Allocate(viskores::BufferSizeType size) const
{
  void* memory = CudaAllocate(size);
  return viskores::cont::internal::BufferInfo(
    viskores::cont::DeviceAdapterTagCuda{}, memory, memory, size, CudaDelete, CudaReallocate);
}

viskores::cont::DeviceAdapterId
DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::GetDevice() const
{
  return viskores::cont::DeviceAdapterTagCuda{};
}

viskores::cont::internal::BufferInfo
DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyHostToDevice(
  const viskores::cont::internal::BufferInfo& src) const
{
  VISKORES_ASSERT(src.GetDevice() == viskores::cont::DeviceAdapterTagUndefined{});

  if (viskores::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()))
  {
    // In the current code structure, we don't know whether this buffer is going to be used
    // for input or output. (Currently, I don't think there is any difference.)
    viskores::cont::cuda::internal::CudaAllocator::PrepareForOutput(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));

    // The provided control pointer is already cuda managed and can be accessed on the device
    // via unified memory. Just shallow copy the pointer.
    return viskores::cont::internal::BufferInfo(src, viskores::cont::DeviceAdapterTagCuda{});
  }
  else
  {
    // Make a new buffer
    viskores::cont::internal::BufferInfo dest = this->Allocate(src.GetSize());

    this->CopyHostToDevice(src, dest);

    return dest;
  }
}

void DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyHostToDevice(
  const viskores::cont::internal::BufferInfo& src,
  const viskores::cont::internal::BufferInfo& dest) const
{
  if (viskores::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()) &&
      src.GetPointer() == dest.GetPointer())
  {
    // In the current code structure, we don't know whether this buffer is going to be used
    // for input or output. (Currently, I don't think there is any difference.)
    viskores::cont::cuda::internal::CudaAllocator::PrepareForOutput(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));

    // The provided pointers are both cuda managed and the same, so the data are already
    // the same.
  }
  else
  {
    viskores::BufferSizeType size = viskores::Min(src.GetSize(), dest.GetSize());

    VISKORES_LOG_F(viskores::cont::LogLevel::MemTransfer,
                   "Copying host --> CUDA dev: %s (%lld bytes)",
                   viskores::cont::GetHumanReadableSize(static_cast<std::size_t>(size)).c_str(),
                   size);

    VISKORES_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                       src.GetPointer(),
                                       static_cast<std::size_t>(size),
                                       hipMemcpyHostToDevice,
                                       hipStreamPerThread));
  }
}


viskores::cont::internal::BufferInfo
DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyDeviceToHost(
  const viskores::cont::internal::BufferInfo& src) const
{
  VISKORES_ASSERT(src.GetDevice() == viskores::cont::DeviceAdapterTagCuda{});

  viskores::cont::internal::BufferInfo dest;

  if (viskores::cont::cuda::internal::CudaAllocator::IsManagedPointer(src.GetPointer()))
  {
    // The provided control pointer is already cuda managed and can be accessed on the host
    // via unified memory. Just shallow copy the pointer.
    viskores::cont::cuda::internal::CudaAllocator::PrepareForControl(
      src.GetPointer(), static_cast<std::size_t>(src.GetSize()));
    dest = viskores::cont::internal::BufferInfo(src, viskores::cont::DeviceAdapterTagUndefined{});

    //In all cases we have possibly multiple async calls queued up in
    //our stream. We need to block on the copy back to control since
    //we don't wanting it accessing memory that hasn't finished
    //being used by the GPU
    viskores::cont::DeviceAdapterAlgorithm<DeviceAdapterTagCuda>::Synchronize();
  }
  else
  {
    // Make a new buffer
    dest = viskores::cont::internal::AllocateOnHost(src.GetSize());

    this->CopyDeviceToHost(src, dest);
  }

  return dest;
}

void DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyDeviceToHost(
  const viskores::cont::internal::BufferInfo& src,
  const viskores::cont::internal::BufferInfo& dest) const
{
  if (viskores::cont::cuda::internal::CudaAllocator::IsManagedPointer(dest.GetPointer()) &&
      src.GetPointer() == dest.GetPointer())
  {
    // The provided pointers are both cuda managed and the same, so the data are already
    // the same.
  }
  else
  {
    viskores::BufferSizeType size = viskores::Min(src.GetSize(), dest.GetSize());

    VISKORES_LOG_F(viskores::cont::LogLevel::MemTransfer,
                   "Copying CUDA dev --> host: %s (%lld bytes)",
                   viskores::cont::GetHumanReadableSize(static_cast<std::size_t>(size)).c_str(),
                   size);

    VISKORES_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                       src.GetPointer(),
                                       static_cast<std::size_t>(size),
                                       hipMemcpyDeviceToHost,
                                       hipStreamPerThread));
  }

  //In all cases we have possibly multiple async calls queued up in
  //our stream. We need to block on the copy back to control since
  //we don't wanting it accessing memory that hasn't finished
  //being used by the GPU
  viskores::cont::DeviceAdapterAlgorithm<DeviceAdapterTagCuda>::Synchronize();
}

viskores::cont::internal::BufferInfo
DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyDeviceToDevice(
  const viskores::cont::internal::BufferInfo& src) const
{
  viskores::cont::internal::BufferInfo dest = this->Allocate(src.GetSize());
  this->CopyDeviceToDevice(src, dest);

  return dest;
}

void DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::CopyDeviceToDevice(
  const viskores::cont::internal::BufferInfo& src,
  const viskores::cont::internal::BufferInfo& dest) const
{
  VISKORES_CUDA_CALL(hipMemcpyAsync(dest.GetPointer(),
                                     src.GetPointer(),
                                     static_cast<std::size_t>(src.GetSize()),
                                     hipMemcpyDeviceToDevice,
                                     hipStreamPerThread));
}

void DeviceAdapterMemoryManager<viskores::cont::DeviceAdapterTagCuda>::DeleteRawPointer(
  void* mem) const
{
  CudaDelete(mem);
};
}
}
} // namespace viskores::cont::internal
