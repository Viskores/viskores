//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <viskores/cont/cuda/internal/testing/Testing.h>

#include <viskores/cont/ArrayHandle.h>
#include <viskores/cont/RuntimeDeviceTracker.h>

#include <viskores/cont/cuda/DeviceAdapterCuda.h>
#include <viskores/cont/cuda/ErrorCuda.h>

#include <viskores/cont/cuda/internal/CudaAllocator.h>
#include <viskores/cont/cuda/internal/testing/Testing.h>

#include <hip/hip_runtime.h>

using viskores::cont::cuda::internal::CudaAllocator;

namespace
{

template <typename ValueType>
ValueType* AllocateManagedPointer(viskores::Id numValues)
{
  void* result;
  VISKORES_CUDA_CALL(
    hipMallocManaged(&result, static_cast<size_t>(numValues) * sizeof(ValueType)));
  // Some sanity checks:
  VISKORES_TEST_ASSERT(CudaAllocator::IsDevicePointer(result),
                       "Allocated pointer is not a device pointer.");
  VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(result),
                       "Allocated pointer is not managed.");
  return static_cast<ValueType*>(result);
}

void DeallocateManagedPointer(void* ptr)
{
  VISKORES_TEST_ASSERT(CudaAllocator::IsDevicePointer(ptr),
                       "Pointer to delete is not device pointer.");
  VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(ptr), "Pointer to delete is not managed.");
  VISKORES_CUDA_CALL(hipFree(ptr));
}

template <typename ValueType>
ValueType* AllocateDevicePointer(viskores::Id numValues)
{
  void* result;
  VISKORES_CUDA_CALL(hipMalloc(&result, static_cast<size_t>(numValues) * sizeof(ValueType)));
  // Some sanity checks:
  VISKORES_TEST_ASSERT(CudaAllocator::IsDevicePointer(result),
                       "Allocated pointer is not a device pointer.");
  VISKORES_TEST_ASSERT(!CudaAllocator::IsManagedPointer(result), "Allocated pointer is managed.");
  return static_cast<ValueType*>(result);
}

void DeallocateDevicePointer(void* ptr)
{
  VISKORES_TEST_ASSERT(CudaAllocator::IsDevicePointer(ptr),
                       "Pointer to delete is not a device pointer.");
  VISKORES_TEST_ASSERT(!CudaAllocator::IsManagedPointer(ptr), "Pointer to delete is managed.");
  VISKORES_CUDA_CALL(hipFree(ptr));
}

template <typename ValueType>
viskores::cont::ArrayHandle<ValueType> CreateArrayHandle(viskores::Id numValues, bool managed)
{
  if (managed)
  {
    return viskores::cont::ArrayHandleBasic<ValueType>(AllocateManagedPointer<ValueType>(numValues),
                                                       numValues,
                                                       [](void* ptr)
                                                       { DeallocateManagedPointer(ptr); });
  }
  else
  {
    return viskores::cont::ArrayHandleBasic<ValueType>(AllocateDevicePointer<ValueType>(numValues),
                                                       numValues,
                                                       viskores::cont::DeviceAdapterTagCuda{},
                                                       [](void* ptr)
                                                       { DeallocateDevicePointer(ptr); });
  }
}

template <typename ValueType>
void TestPrepareForInput(bool managed)
{
  viskores::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  viskores::cont::Token token;
  auto portal = handle.PrepareForInput(viskores::cont::DeviceAdapterTagCuda(), token);
  const void* execArray = portal.GetIteratorBegin();
  VISKORES_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForInput.");
  if (managed)
  {
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray));
  }
  token.DetachFromAll();

  VISKORES_TEST_ASSERT(handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}),
                       "No execution array after PrepareForInput.");
  if (managed)
  {
    const void* contArray = handle.ReadPortal().GetIteratorBegin();
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray), "Control array unmanaged.");
    VISKORES_TEST_ASSERT(execArray == contArray, "PrepareForInput managed arrays not shared.");
  }
}

template <typename ValueType>
void TestPrepareForInPlace(bool managed)
{
  viskores::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  viskores::cont::Token token;
  auto portal = handle.PrepareForInPlace(viskores::cont::DeviceAdapterTagCuda(), token);
  const void* execArray = portal.GetIteratorBegin();
  VISKORES_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForInPlace.");
  if (managed)
  {
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray));
  }
  token.DetachFromAll();

  VISKORES_TEST_ASSERT(!handle.IsOnHost(), "Control array still exists after PrepareForInPlace.");
  VISKORES_TEST_ASSERT(handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}),
                       "No execution array after PrepareForInPlace.");
  if (managed)
  {
    const void* contArray = handle.ReadPortal().GetIteratorBegin();
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray), "Control array unmanaged.");
    VISKORES_TEST_ASSERT(execArray == contArray, "PrepareForInPlace managed arrays not shared.");
  }
}

template <typename ValueType>
void TestPrepareForOutput(bool managed)
{
  // Should reuse a managed control pointer if buffer is large enough.
  viskores::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  viskores::cont::Token token;
  auto portal = handle.PrepareForOutput(32, viskores::cont::DeviceAdapterTagCuda(), token);
  const void* execArray = portal.GetIteratorBegin();
  VISKORES_TEST_ASSERT(execArray != nullptr, "No execution array after PrepareForOutput.");
  if (managed)
  {
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray));
  }
  token.DetachFromAll();

  VISKORES_TEST_ASSERT(!handle.IsOnHost(), "Control array still exists after PrepareForOutput.");
  VISKORES_TEST_ASSERT(handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}),
                       "No execution array after PrepareForOutput.");
  if (managed)
  {
    const void* contArray = handle.ReadPortal().GetIteratorBegin();
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray), "Control array unmanaged.");
    VISKORES_TEST_ASSERT(execArray == contArray, "PrepareForOutput managed arrays not shared.");
  }
}

template <typename ValueType>
void TestReleaseResourcesExecution(bool managed)
{
  viskores::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  viskores::cont::Token token;
  auto portal = handle.PrepareForInput(viskores::cont::DeviceAdapterTagCuda(), token);
  const void* origArray = portal.GetIteratorBegin();
  token.DetachFromAll();

  handle.ReleaseResourcesExecution();

  VISKORES_TEST_ASSERT(handle.IsOnHost(),
                       "Control array does not exist after ReleaseResourcesExecution.");
  VISKORES_TEST_ASSERT(!handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}),
                       "Execution array still exists after ReleaseResourcesExecution.");

  if (managed)
  {
    const void* contArray = handle.ReadPortal().GetIteratorBegin();
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray), "Control array unmanaged.");
    VISKORES_TEST_ASSERT(origArray == contArray, "Managed arrays not shared.");
  }
}

template <typename ValueType>
void TestRoundTrip(bool managed)
{
  viskores::cont::ArrayHandle<ValueType> handle = CreateArrayHandle<ValueType>(32, managed);
  const void* origExecArray;
  {
    viskores::cont::Token token;
    auto portal = handle.PrepareForOutput(32, viskores::cont::DeviceAdapterTagCuda(), token);
    origExecArray = portal.GetIteratorBegin();
  }

  VISKORES_TEST_ASSERT(!handle.IsOnHost());
  VISKORES_TEST_ASSERT(handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}));

  const void* contArray;
  {
    auto portal = handle.WritePortal();
    contArray = portal.GetIteratorBegin();
  }

  VISKORES_TEST_ASSERT(handle.IsOnHost());
  VISKORES_TEST_ASSERT(!handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}));
  if (managed)
  {
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(contArray));
    VISKORES_TEST_ASSERT(contArray == origExecArray);
  }

  const void* execArray;
  {
    viskores::cont::Token token;
    auto portal = handle.PrepareForInput(viskores::cont::DeviceAdapterTagCuda(), token);
    execArray = portal.GetIteratorBegin();
  }

  VISKORES_TEST_ASSERT(handle.IsOnHost());
  VISKORES_TEST_ASSERT(handle.IsOnDevice(viskores::cont::DeviceAdapterTagCuda{}));
  if (managed)
  {
    VISKORES_TEST_ASSERT(CudaAllocator::IsManagedPointer(execArray));
    VISKORES_TEST_ASSERT(execArray == contArray);
  }
}

template <typename ValueType>
void DoTests()
{
  TestPrepareForInput<ValueType>(false);
  TestPrepareForInPlace<ValueType>(false);
  TestPrepareForOutput<ValueType>(false);
  TestReleaseResourcesExecution<ValueType>(false);
  TestRoundTrip<ValueType>(false);


  // If this device does not support managed memory, skip the managed tests.
  if (!CudaAllocator::UsingManagedMemory())
  {
    std::cerr << "Skipping some tests -- device does not support managed memory.\n";
  }
  else
  {
    TestPrepareForInput<ValueType>(true);
    TestPrepareForInPlace<ValueType>(true);
    TestPrepareForOutput<ValueType>(true);
    TestReleaseResourcesExecution<ValueType>(true);
    TestRoundTrip<ValueType>(true);
  }
}

struct ArgToTemplateType
{
  template <typename ValueType>
  void operator()(ValueType) const
  {
    DoTests<ValueType>();
  }
};

void Launch()
{
  using Types = viskores::List<viskores::UInt8,
                               viskores::Vec<viskores::UInt8, 3>,
                               viskores::Float32,
                               viskores::Vec<viskores::Float32, 4>,
                               viskores::Float64,
                               viskores::Vec<viskores::Float64, 4>>;
  viskores::testing::Testing::TryTypes(ArgToTemplateType(), Types());
}

} // end anon namespace

int UnitTestCudaShareUserProvidedManagedMemory(int argc, char* argv[])
{
  auto& tracker = viskores::cont::GetRuntimeDeviceTracker();
  tracker.ForceDevice(viskores::cont::DeviceAdapterTagCuda{});
  int ret = viskores::cont::testing::Testing::Run(Launch, argc, argv);
  return viskores::cont::cuda::internal::Testing::CheckCudaBeforeExit(ret);
}
