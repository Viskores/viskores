#include "hip/hip_runtime.h"
//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <viskores/cont/RuntimeDeviceTracker.h>
#include <viskores/cont/cuda/DeviceAdapterCuda.h>

#include <viskores/worklet/DispatcherMapField.h>
#include <viskores/worklet/WorkletMapField.h>

#include <viskores/cont/testing/Testing.h>

#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

VISKORES_THIRDPARTY_PRE_INCLUDE
#include "hiprand/hiprand_kernel.h"
VISKORES_THIRDPARTY_POST_INCLUDE

namespace
{

struct TriggerICE : public viskores::worklet::WorkletMapField
{
  using ControlSignature = void(FieldIn, FieldIn, FieldOut);
  using ExecutionSignature = _3(_1, _2, WorkIndex);

#ifdef VISKORES_CUDA_DEVICE_PASS
  template <class ValueType>
  __device__ ValueType operator()(const ValueType& bad,
                                  const ValueType& sane,
                                  const viskores::Id sequenceId) const
  {

    hiprandState_t state;
    //Each thread uses same seed but different sequence numbers
    hiprand_init(42, sequenceId, 0, &state);

    int signBad = viskores::SignBit(bad);
    int signGood = viskores::SignBit(bad);

    viskores::Vec<ValueType, 3> coord = { viskores::Abs(bad * sane),
                                          bad * sane + (ValueType)signBad,
                                          bad * sane + (ValueType)signGood };

    for (int i = 0; i < 10; ++i)
    {
      for (int j = 0; j < 3; ++j)
      {
        if (viskores::IsNan(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 5.0f;
          coord[j] = viskores::Sqrt(viskores::Dot(coord, coord));
          if (coord[j] <= 1.0f)
          {
            coord[j] += 1.0f;
          }
        }
        if (viskores::IsInf(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 8.0f;
          coord[j] = viskores::Tan(viskores::Cos(viskores::Dot(coord, coord)));
        }
      }
    }
    return coord[0] * 4.0f + coord[1] * 4.0f + coord[2] * 4.0f;
  }
#else
  template <class ValueType>
  ValueType operator()(const ValueType& bad,
                       const ValueType& sane,
                       const viskores::Id sequenceId) const
  {
    return bad + sane * static_cast<ValueType>(sequenceId);
  }
#endif
};

//-----------------------------------------------------------------------------
template <typename Device>
void RunEdgeCases()
{
  std::cout << "Testing complicated worklets that can cause NVCC to ICE." << std::endl;
  //When running CUDA on unsupported hardware we find that IsInf, IsNan, and
  //SignBit can cause the CUDA compiler to crash. This test is a consistent
  //way to detect this.
  //
  //The way it works is we generate all kinds of nasty floating point values
  //such as signaling Nan, quiet Nan, other Nans, +Inf, -Inf, -0, +0, a collection of
  //denormal numbers, and the min and max float values
  //and than a random collection of values from normal float space. We combine this
  //array which we will call 'bad' with another input array which we will call 'sane',
  //We than execute a worklet that takes values stored in 'bad' and 'sane' that does
  //some computation that takes into account the results of IsInf, IsNan, and
  //SignBit
  const viskores::Id desired_size = 2048;
  std::vector<float> sanevalues;
  std::vector<float> badvalues = { std::numeric_limits<float>::signaling_NaN(),
                                   std::numeric_limits<float>::quiet_NaN(),
                                   std::nanf("1"),
                                   std::nanf("4200042"),
                                   std::numeric_limits<float>::infinity(),
                                   std::numeric_limits<float>::infinity() * -1,
                                   0.0f,
                                   -0.0f,
                                   std::numeric_limits<float>::denorm_min(),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f),
                                   std::numeric_limits<float>::denorm_min() *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f) *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::numeric_limits<float>::lowest(),
                                   std::numeric_limits<float>::min(),
                                   std::numeric_limits<float>::max() };
  const std::size_t bad_size = badvalues.size();
  const viskores::Id bad_size_as_id = static_cast<viskores::Id>(bad_size);

  badvalues.reserve(desired_size);
  sanevalues.reserve(desired_size);

  //construct a random number generator
  std::mt19937 rng;
  std::uniform_real_distribution<float> range(-1.0f, 1.0f);

  // now add in some random numbers to the bad values
  for (std::size_t i = 0; i < desired_size - bad_size; ++i)
  {
    badvalues.push_back(range(rng));
  }
  for (std::size_t i = 0; i < desired_size; ++i)
  {
    sanevalues.push_back(range(rng));
  }

  auto bad = viskores::cont::make_ArrayHandle(badvalues, viskores::CopyFlag::On);
  auto sane = viskores::cont::make_ArrayHandle(sanevalues, viskores::CopyFlag::On);
  decltype(sane) result;
  viskores::worklet::DispatcherMapField<TriggerICE> dispatcher;
  dispatcher.SetDevice(Device());
  dispatcher.Invoke(bad, sane, result);

  auto portal = result.ReadPortal();

  //the first 6 values should be nan
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(0)), "Value should be NaN.");
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(1)), "Value should be NaN.");
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(2)), "Value should be NaN.");
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(3)), "Value should be NaN.");
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(4)), "Value should be NaN.");
  VISKORES_TEST_ASSERT(viskores::IsNan(portal.Get(5)), "Value should be NaN.");

  for (viskores::Id i = bad_size_as_id; i < desired_size; ++i)
  { //The rest of the values shouldn't be Nan or Inf
    auto v = portal.Get(i);
    const bool valid = !viskores::IsNan(v) && !viskores::IsInf(v);
    VISKORES_TEST_ASSERT(valid, "value shouldn't be NaN or INF");
  }
}

} //namespace

int UnitTestCudaMathEdgeCases(int argc, char* argv[])
{
  auto& tracker = viskores::cont::GetRuntimeDeviceTracker();
  tracker.ForceDevice(viskores::cont::DeviceAdapterTagCuda{});

  return viskores::cont::testing::Testing::Run(
    RunEdgeCases<viskores::cont::DeviceAdapterTagCuda>, argc, argv);
}
