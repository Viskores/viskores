#include "hip/hip_runtime.h"
//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <viskores/cont/RuntimeDeviceTracker.h>
#include <viskores/cont/cuda/DeviceAdapterCuda.h>
#include <viskores/cont/testing/TestingRuntimeDeviceConfiguration.h>

namespace internal = viskores::cont::internal;

namespace viskores
{
namespace cont
{
namespace testing
{

template <>
VISKORES_CONT void
TestingRuntimeDeviceConfiguration<viskores::cont::DeviceAdapterTagCuda>::TestRuntimeConfig()
{
  auto deviceOptions = TestingRuntimeDeviceConfiguration::DefaultInitializeConfigOptions();
  int numDevices = 0;
  VISKORES_CUDA_CALL(hipGetDeviceCount(&numDevices));
  viskores::Id selectedDevice = numDevices > 0 ? numDevices - 1 : 0;
  deviceOptions.ViskoresDeviceInstance.SetOption(selectedDevice);
  auto& config =
    RuntimeDeviceInformation{}.GetRuntimeConfiguration(DeviceAdapterTagCuda(), deviceOptions);
  viskores::Id setDevice;
  VISKORES_TEST_ASSERT(config.GetDeviceInstance(setDevice) ==
                         internal::RuntimeDeviceConfigReturnCode::SUCCESS,
                       "Failed to get device instance");
  VISKORES_TEST_ASSERT(setDevice == selectedDevice,
                       "RTC's setDevice != selectedDevice cuda direct! " +
                         std::to_string(setDevice) + " != " + std::to_string(selectedDevice));
  viskores::Id maxDevices;
  VISKORES_TEST_ASSERT(config.GetMaxDevices(maxDevices) ==
                         internal::RuntimeDeviceConfigReturnCode::SUCCESS,
                       "Failed to get max devices");
  VISKORES_TEST_ASSERT(maxDevices == numDevices,
                       "RTC's maxDevices != numDevices cuda direct! " + std::to_string(maxDevices) +
                         " != " + std::to_string(numDevices));
  std::vector<hipDeviceProp_t> cudaProps;
  dynamic_cast<internal::RuntimeDeviceConfiguration<viskores::cont::DeviceAdapterTagCuda>&>(config)
    .GetCudaDeviceProp(cudaProps);
  VISKORES_TEST_ASSERT(maxDevices == static_cast<viskores::Id>(cudaProps.size()),
                       "CudaProp's size != maxDevices! " + std::to_string(cudaProps.size()) +
                         " != " + std::to_string(maxDevices));
}

} // namespace viskores::cont::testing
} // namespace viskores::cont
} // namespace viskores

int UnitTestCudaRuntimeDeviceConfiguration(int argc, char* argv[])
{
  return viskores::cont::testing::TestingRuntimeDeviceConfiguration<
    viskores::cont::DeviceAdapterTagCuda>::Run(argc, argv);
}
