#include "hip/hip_runtime.h"
//============================================================================
//  The contents of this file are covered by the Viskores license. See
//  LICENSE.txt for details.
//
//  By contributing to this file, all contributors agree to the Developer
//  Certificate of Origin Version 1.1 (DCO 1.1) as stated in DCO.txt.
//============================================================================

//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================
#include <viskores/testing/Testing.h>

#include <viskores/cont/cuda/DeviceAdapterCuda.h>

#include <viskores/exec/FunctorBase.h>
#include <viskores/exec/arg/BasicArg.h>
#include <viskores/exec/arg/ThreadIndicesBasic.h>
#include <viskores/exec/cuda/internal/TaskStrided.h>

#include <viskores/StaticAssert.h>

#include <viskores/internal/FunctionInterface.h>
#include <viskores/internal/Invocation.h>

#if defined(VISKORES_MSVC)
#pragma warning(push)
#pragma warning(disable : 4068) //unknown pragma
#endif

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
// Disable warning "declared but never referenced"
// This file produces several false-positive warnings
// Eg: TestExecObject::TestExecObject, MyOutputToInputMapPortal::Get,
//     TestWorkletProxy::operator()
#pragma push
#if (CUDART_VERSION >= 11050)
#pragma nv_diag_suppress 177
#else
#pragma diag_suppress 177
#endif

#endif

namespace
{

struct TestExecObject
{
  using PortalType = viskores::cont::ArrayHandle<viskores::Id>::WritePortalType;

  VISKORES_EXEC_CONT
  TestExecObject(PortalType portal)
    : Portal(portal)
  {
  }

  PortalType Portal;
};

struct MyOutputToInputMapPortal
{
  using ValueType = viskores::Id;
  VISKORES_EXEC_CONT
  viskores::Id Get(viskores::Id index) const { return index; }
};

struct MyVisitArrayPortal
{
  using ValueType = viskores::IdComponent;
  VISKORES_EXEC_CONT
  viskores::IdComponent Get(viskores::Id) const { return 1; }
};

struct MyThreadToOutputMapPortal
{
  using ValueType = viskores::Id;
  VISKORES_EXEC_CONT
  viskores::Id Get(viskores::Id index) const { return index; }
};

struct TestFetchTagInput
{
};
struct TestFetchTagOutput
{
};

// Missing TransportTag, but we are not testing that so we can leave it out.
struct TestControlSignatureTagInput
{
  using FetchTag = TestFetchTagInput;
};
struct TestControlSignatureTagOutput
{
  using FetchTag = TestFetchTagOutput;
};

} // anonymous namespace

namespace viskores
{
namespace exec
{
namespace arg
{

template <>
struct Fetch<TestFetchTagInput, viskores::exec::arg::AspectTagDefault, TestExecObject>
{
  using ValueType = viskores::Id;

  VISKORES_EXEC
  ValueType Load(const viskores::exec::arg::ThreadIndicesBasic& indices,
                 const TestExecObject& execObject) const
  {
    return execObject.Portal.Get(indices.GetInputIndex()) + 10 * indices.GetInputIndex();
  }

  VISKORES_EXEC
  void Store(const viskores::exec::arg::ThreadIndicesBasic&, const TestExecObject&, ValueType) const
  {
    // No-op
  }
};

template <>
struct Fetch<TestFetchTagOutput, viskores::exec::arg::AspectTagDefault, TestExecObject>
{
  using ValueType = viskores::Id;

  VISKORES_EXEC
  ValueType Load(const viskores::exec::arg::ThreadIndicesBasic&, const TestExecObject&) const
  {
    // No-op
    return ValueType();
  }

  VISKORES_EXEC
  void Store(const viskores::exec::arg::ThreadIndicesBasic& indices,
             const TestExecObject& execObject,
             ValueType value) const
  {
    execObject.Portal.Set(indices.GetOutputIndex(), value + 20 * indices.GetOutputIndex());
  }
};
}
}
} // viskores::exec::arg

namespace
{

using TestControlSignature = void(TestControlSignatureTagInput, TestControlSignatureTagOutput);
using TestControlInterface = viskores::internal::FunctionInterface<TestControlSignature>;

using TestExecutionSignature1 = void(viskores::exec::arg::BasicArg<1>,
                                     viskores::exec::arg::BasicArg<2>);
using TestExecutionInterface1 = viskores::internal::FunctionInterface<TestExecutionSignature1>;

using TestExecutionSignature2 = viskores::exec::arg::BasicArg<2>(viskores::exec::arg::BasicArg<1>);
using TestExecutionInterface2 = viskores::internal::FunctionInterface<TestExecutionSignature2>;

using ExecutionParameterInterface =
  viskores::internal::FunctionInterface<void(TestExecObject, TestExecObject)>;

using InvocationType1 = viskores::internal::Invocation<ExecutionParameterInterface,
                                                       TestControlInterface,
                                                       TestExecutionInterface1,
                                                       1,
                                                       MyOutputToInputMapPortal,
                                                       MyVisitArrayPortal,
                                                       MyThreadToOutputMapPortal>;

using InvocationType2 = viskores::internal::Invocation<ExecutionParameterInterface,
                                                       TestControlInterface,
                                                       TestExecutionInterface2,
                                                       1,
                                                       MyOutputToInputMapPortal,
                                                       MyVisitArrayPortal,
                                                       MyThreadToOutputMapPortal>;

template <typename TaskType>
static __global__ void ScheduleTaskStrided(TaskType task, viskores::Id start, viskores::Id end)
{

  const viskores::Id index = blockIdx.x * blockDim.x + threadIdx.x;
  const viskores::Id inc = blockDim.x * gridDim.x;
  if (index >= start && index < end)
  {
    task(index, end, inc);
  }
}

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletProxy : viskores::exec::FunctorBase
{
  VISKORES_EXEC
  void operator()(viskores::Id input, viskores::Id& output) const { output = input + 100; }

  VISKORES_EXEC
  viskores::Id operator()(viskores::Id input) const { return input + 200; }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType>
  VISKORES_EXEC viskores::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&) const
  {
    viskores::Id outIndex = threadToOut.Get(threadIndex);
    return viskores::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex);
  }
};

#define ERROR_MESSAGE "Expected worklet error."

// Not a full worklet, but provides operators that we expect in a worklet.
struct TestWorkletErrorProxy : viskores::exec::FunctorBase
{
  VISKORES_EXEC
  void operator()(viskores::Id, viskores::Id) const { this->RaiseError(ERROR_MESSAGE); }

  template <typename T,
            typename OutToInArrayType,
            typename VisitArrayType,
            typename ThreadToOutArrayType,
            typename InputDomainType>
  VISKORES_EXEC viskores::exec::arg::ThreadIndicesBasic GetThreadIndices(
    const T& threadIndex,
    const OutToInArrayType& outToIn,
    const VisitArrayType& visit,
    const ThreadToOutArrayType& threadToOut,
    const InputDomainType&) const
  {
    viskores::Id outIndex = threadToOut.Get(threadIndex);
    return viskores::exec::arg::ThreadIndicesBasic(
      threadIndex, outToIn.Get(outIndex), visit.Get(outIndex), outIndex);
  }
};

// Check behavior of InvocationToFetch helper class.

VISKORES_STATIC_ASSERT(
  (std::is_same<
    viskores::exec::internal::detail::
      InvocationToFetch<viskores::exec::arg::ThreadIndicesBasic, InvocationType1, 1>::type,
    viskores::exec::arg::Fetch<TestFetchTagInput,
                               viskores::exec::arg::AspectTagDefault,
                               TestExecObject>>::type::value));

VISKORES_STATIC_ASSERT(
  (std::is_same<
    viskores::exec::internal::detail::
      InvocationToFetch<viskores::exec::arg::ThreadIndicesBasic, InvocationType1, 2>::type,
    viskores::exec::arg::Fetch<TestFetchTagOutput,
                               viskores::exec::arg::AspectTagDefault,
                               TestExecObject>>::type::value));

VISKORES_STATIC_ASSERT(
  (std::is_same<
    viskores::exec::internal::detail::
      InvocationToFetch<viskores::exec::arg::ThreadIndicesBasic, InvocationType2, 0>::type,
    viskores::exec::arg::Fetch<TestFetchTagOutput,
                               viskores::exec::arg::AspectTagDefault,
                               TestExecObject>>::type::value));

template <typename DeviceAdapter>
void TestNormalFunctorInvoke()
{
  std::cout << "Testing normal worklet invoke." << std::endl;

  viskores::cont::Token token;

  viskores::Id inputTestValues[3] = { 5, 5, 6 };

  viskores::cont::ArrayHandle<viskores::Id> input =
    viskores::cont::make_ArrayHandle(inputTestValues, 3, viskores::CopyFlag::Off);
  viskores::cont::ArrayHandle<viskores::Id> output;

  viskores::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    viskores::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
      TestExecObject(output.PrepareForOutput(3, DeviceAdapter(), token)));

  std::cout << "  Try void return." << std::endl;
  TestWorkletProxy worklet;
  InvocationType1 invocation1(execObjects);

  using TaskTypes = typename viskores::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task1 = TaskTypes::MakeTask(worklet, invocation1, viskores::Id());

  ScheduleTaskStrided<decltype(task1)><<<32, 256>>>(task1, 1, 2);
  hipDeviceSynchronize();
  token.DetachFromAll();
  input.SyncControlArray();
  output.SyncControlArray();

  VISKORES_TEST_ASSERT(inputTestValues[1] == 5, "Input value changed.");
  VISKORES_TEST_ASSERT(output.ReadPortal().Get(1) == inputTestValues[1] + 100 + 30,
                       "Output value not set right.");

  std::cout << "  Try return value." << std::endl;

  execObjects = viskores::internal::make_FunctionInterface<void>(
    TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
    TestExecObject(output.PrepareForOutput(3, DeviceAdapter(), token)));

  InvocationType2 invocation2(execObjects);

  using TaskTypes = typename viskores::cont::DeviceTaskTypes<DeviceAdapter>;
  auto task2 = TaskTypes::MakeTask(worklet, invocation2, viskores::Id());

  ScheduleTaskStrided<decltype(task2)><<<32, 256>>>(task2, 2, 3);
  hipDeviceSynchronize();
  token.DetachFromAll();
  input.SyncControlArray();
  output.SyncControlArray();

  VISKORES_TEST_ASSERT(inputTestValues[2] == 6, "Input value changed.");
  VISKORES_TEST_ASSERT(output.ReadPortal().Get(2) == inputTestValues[2] + 200 + 30 * 2,
                       "Output value not set right.");
}

template <typename DeviceAdapter>
void TestErrorFunctorInvoke()
{
  std::cout << "Testing invoke with an error raised in the worklet." << std::endl;

  viskores::cont::Token token;

  viskores::Id inputTestValue = 5;
  viskores::Id outputTestValue = static_cast<viskores::Id>(0xDEADDEAD);

  viskores::cont::ArrayHandle<viskores::Id> input =
    viskores::cont::make_ArrayHandle(&inputTestValue, 1, viskores::CopyFlag::Off);
  viskores::cont::ArrayHandle<viskores::Id> output =
    viskores::cont::make_ArrayHandle(&outputTestValue, 1, viskores::CopyFlag::Off);

  viskores::internal::FunctionInterface<void(TestExecObject, TestExecObject)> execObjects =
    viskores::internal::make_FunctionInterface<void>(
      TestExecObject(input.PrepareForInPlace(DeviceAdapter(), token)),
      TestExecObject(output.PrepareForInPlace(DeviceAdapter(), token)));

  using TaskStrided1 = viskores::exec::cuda::internal::
    TaskStrided1D<TestWorkletErrorProxy, InvocationType1, viskores::cont::internal::HintList<>>;
  TestWorkletErrorProxy worklet;
  InvocationType1 invocation(execObjects);

  using TaskTypes = typename viskores::cont::DeviceTaskTypes<DeviceAdapter>;
  using Algorithm = viskores::cont::DeviceAdapterAlgorithm<DeviceAdapter>;

  auto task = TaskTypes::MakeTask(worklet, invocation, viskores::Id());

  auto errorArray = Algorithm::GetPinnedErrorArray();
  viskores::exec::internal::ErrorMessageBuffer errorMessage(errorArray.DevicePtr, errorArray.Size);
  task.SetErrorMessageBuffer(errorMessage);

  ScheduleTaskStrided<decltype(task)><<<32, 256>>>(task, 1, 2);
  hipDeviceSynchronize();

  VISKORES_TEST_ASSERT(errorMessage.IsErrorRaised(), "Error not raised correctly.");
  VISKORES_TEST_ASSERT(errorArray.HostPtr == std::string(ERROR_MESSAGE),
                       "Got wrong error message.");
}

template <typename DeviceAdapter>
void TestTaskStrided()
{
  TestNormalFunctorInvoke<DeviceAdapter>();
  TestErrorFunctorInvoke<DeviceAdapter>();
}

} // anonymous namespace

int UnitTestTaskStrided(int argc, char* argv[])
{
  return viskores::testing::Testing::Run(
    TestTaskStrided<viskores::cont::DeviceAdapterTagCuda>, argc, argv);
}

#if defined(__NVCC__) && defined(__CUDACC_VER_MAJOR__)
#pragma pop
#endif

#if defined(VISKORES_MSVC)
#pragma warning(pop)
#endif
