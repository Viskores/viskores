//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================
#include <vtkm/cont/cuda/internal/CudaAllocator.h>
#include <vtkm/cont/cuda/internal/ExecutionArrayInterfaceBasicCuda.h>

#include <vtkm/cont/Logging.h>

using vtkm::cont::cuda::internal::CudaAllocator;

namespace vtkm
{
namespace cont
{
namespace internal
{

DeviceAdapterId ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::GetDeviceId() const
{
  return DeviceAdapterTagCuda{};
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Allocate(TypelessExecutionArray& execArray,
                                                                  vtkm::Id numberOfValues,
                                                                  vtkm::UInt64 sizeOfValue) const
{
  const vtkm::UInt64 numBytes = static_cast<vtkm::UInt64>(numberOfValues) * sizeOfValue;
  // Detect if we can reuse a device-accessible pointer from the control env:
  if (CudaAllocator::IsDevicePointer(execArray.ArrayControl))
  {
    const vtkm::UInt64 managedCapacity =
      static_cast<vtkm::UInt64>(static_cast<const char*>(execArray.ArrayControlCapacity) -
                                static_cast<const char*>(execArray.ArrayControl));
    if (managedCapacity >= numBytes)
    {
      if (execArray.Array && execArray.Array != execArray.ArrayControl)
      {
        this->Free(execArray);
      }

      execArray.Array = const_cast<void*>(execArray.ArrayControl);
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      execArray.ArrayCapacity = const_cast<void*>(execArray.ArrayControlCapacity);
      return;
    }
  }

  if (execArray.Array != nullptr)
  {
    const vtkm::UInt64 cap = static_cast<vtkm::UInt64>(static_cast<char*>(execArray.ArrayCapacity) -
                                                       static_cast<char*>(execArray.Array));

    if (cap < numBytes)
    { // Current allocation too small -- free & realloc
      this->Free(execArray);
    }
    else
    { // Reuse buffer if possible:
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      return;
    }
  }

  VTKM_ASSERT(execArray.Array == nullptr);

  // Attempt to allocate:
  try
  {
    // Cast to char* so that the pointer math below will work.
    char* tmp = static_cast<char*>(CudaAllocator::Allocate(static_cast<size_t>(numBytes)));
    execArray.Array = tmp;
    execArray.ArrayEnd = tmp + numBytes;
    execArray.ArrayCapacity = tmp + numBytes;
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << numBytes << " bytes on device: " << error.what();
    throw vtkm::cont::ErrorBadAllocation(err.str());
  }

  // If we just allocated managed cuda memory and don't a host memory pointer
  // we can share out managed memory. This allows for the use case of where we
  // first allocate on CUDA and than want to use it on the host
  if (CudaAllocator::IsManagedPointer(execArray.Array) && execArray.ArrayControl == nullptr)
  {
    this->ControlStorage.SetBasePointer(
      execArray.Array, numberOfValues, sizeOfValue, [](void* ptr) { CudaAllocator::Free(ptr); });
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Free(
  TypelessExecutionArray& execArray) const
{
  // If we're sharing a device-accessible pointer between control/exec, don't
  // actually free it -- just null the pointers here:
  if (execArray.Array == execArray.ArrayControl &&
      CudaAllocator::IsDevicePointer(execArray.ArrayControl))
  {
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
    return;
  }

  if (execArray.Array != nullptr)
  {
    const vtkm::UInt64 cap = static_cast<vtkm::UInt64>(static_cast<char*>(execArray.ArrayCapacity) -
                                                       static_cast<char*>(execArray.Array));

    CudaAllocator::FreeDeferred(execArray.Array, cap);
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyFromControl(
  const void* controlPtr,
  void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  // Do nothing if we're sharing a device-accessible pointer between control and
  // execution:
  if (controlPtr == executionPtr && CudaAllocator::IsDevicePointer(controlPtr))
  {
    CudaAllocator::PrepareForInput(executionPtr, numBytes);
    return;
  }

  VTKM_LOG_F(vtkm::cont::LogLevel::MemTransfer,
             "Copying host --> CUDA dev: %s (%llu bytes)",
             vtkm::cont::GetHumanReadableSize(numBytes).c_str(),
             numBytes);

  VTKM_CUDA_CALL(hipMemcpyAsync(executionPtr,
                                 controlPtr,
                                 static_cast<std::size_t>(numBytes),
                                 hipMemcpyHostToDevice,
                                 hipStreamPerThread));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyToControl(const void* executionPtr,
                                                                       void* controlPtr,
                                                                       vtkm::UInt64 numBytes) const
{
  // Do nothing if we're sharing a cuda managed pointer between control and execution:
  if (controlPtr == executionPtr && CudaAllocator::IsDevicePointer(controlPtr))
  {
    // If we're trying to copy a shared, non-managed device pointer back to
    // control throw an exception -- the pointer cannot be read from control,
    // so this operation is invalid.
    if (!CudaAllocator::IsManagedPointer(controlPtr))
    {
      throw vtkm::cont::ErrorBadValue(
        "Control pointer is a CUDA device pointer that does not supported managed access.");
    }

    // If it is managed, just return and let CUDA handle the migration for us.
    CudaAllocator::PrepareForControl(controlPtr, numBytes);
  }
  else
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemTransfer,
               "Copying CUDA dev --> host: %s (%llu bytes)",
               vtkm::cont::GetHumanReadableSize(numBytes).c_str(),
               numBytes);

    VTKM_CUDA_CALL(hipMemcpyAsync(controlPtr,
                                   executionPtr,
                                   static_cast<std::size_t>(numBytes),
                                   hipMemcpyDeviceToHost,
                                   hipStreamPerThread));
  }

  //In all cases we have possibly multiple async calls queued up in
  //our stream. We need to block on the copy back to control since
  //we don't wanting it accessing memory that hasn't finished
  //being used by the GPU
  vtkm::cont::DeviceAdapterAlgorithm<DeviceAdapterTagCuda>::Synchronize();
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForRead(
  const void* vtkmNotUsed(controlPtr),
  const void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForInput(executionPtr, static_cast<size_t>(numBytes));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForWrite(
  const void* vtkmNotUsed(controlPtr),
  const void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForOutput(executionPtr, static_cast<size_t>(numBytes));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForReadWrite(
  const void* vtkmNotUsed(controlPtr),
  const void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForInPlace(executionPtr, static_cast<size_t>(numBytes));
}


} // end namespace internal
}
} // end vtkm::cont
