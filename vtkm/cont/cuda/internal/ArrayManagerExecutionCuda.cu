//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#define vtk_m_cont_cuda_internal_ArrayManagerExecutionCuda_cu

#include <vtkm/cont/cuda/internal/ArrayManagerExecutionCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>

using vtkm::cont::cuda::internal::CudaAllocator;

namespace vtkm
{
namespace cont
{
namespace internal
{

ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::ExecutionArrayInterfaceBasic(
  StorageBasicBase& storage)
  : Superclass(storage)
{
}

DeviceAdapterId ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::GetDeviceId() const
{
  return VTKM_DEVICE_ADAPTER_CUDA;
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Allocate(TypelessExecutionArray& execArray,
                                                                  vtkm::UInt64 numBytes) const
{
  // Detect if we can reuse a device-accessible pointer from the control env:
  if (CudaAllocator::IsDevicePointer(execArray.ArrayControl))
  {
    const vtkm::UInt64 managedCapacity =
      static_cast<vtkm::UInt64>(static_cast<const char*>(execArray.ArrayControlCapacity) -
                                static_cast<const char*>(execArray.ArrayControl));
    if (managedCapacity >= numBytes)
    {
      if (execArray.Array && execArray.Array != execArray.ArrayControl)
      {
        this->Free(execArray);
      }

      execArray.Array = const_cast<void*>(execArray.ArrayControl);
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      execArray.ArrayCapacity = const_cast<void*>(execArray.ArrayControlCapacity);
      return;
    }
  }

  if (execArray.Array != nullptr)
  {
    const vtkm::UInt64 cap = static_cast<vtkm::UInt64>(static_cast<char*>(execArray.ArrayCapacity) -
                                                       static_cast<char*>(execArray.Array));

    if (cap < numBytes)
    { // Current allocation too small -- free & realloc
      this->Free(execArray);
    }
    else
    { // Reuse buffer if possible:
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      return;
    }
  }

  VTKM_ASSERT(execArray.Array == nullptr);

  // Attempt to allocate:
  try
  {
    // Cast to char* so that the pointer math below will work.
    char* tmp = static_cast<char*>(CudaAllocator::Allocate(static_cast<size_t>(numBytes)));
    execArray.Array = tmp;
    execArray.ArrayEnd = tmp + numBytes;
    execArray.ArrayCapacity = tmp + numBytes;
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << numBytes << " bytes on device: " << error.what();
    throw vtkm::cont::ErrorBadAllocation(err.str());
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Free(
  TypelessExecutionArray& execArray) const
{
  // If we're sharing a device-accessible pointer between control/exec, don't
  // actually free it -- just null the pointers here:
  if (execArray.Array == execArray.ArrayControl &&
      CudaAllocator::IsDevicePointer(execArray.ArrayControl))
  {
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
    return;
  }

  if (execArray.Array != nullptr)
  {
    CudaAllocator::Free(execArray.Array);
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyFromControl(
  const void* controlPtr,
  void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  // Do nothing if we're sharing a device-accessible pointer between control and
  // execution:
  if (controlPtr == executionPtr && CudaAllocator::IsDevicePointer(controlPtr))
  {
    CudaAllocator::PrepareForInput(executionPtr, numBytes);
    return;
  }

  VTKM_CUDA_CALL(hipMemcpyAsync(executionPtr,
                                 controlPtr,
                                 static_cast<std::size_t>(numBytes),
                                 hipMemcpyHostToDevice,
                                 hipStreamPerThread));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyToControl(const void* executionPtr,
                                                                       void* controlPtr,
                                                                       vtkm::UInt64 numBytes) const
{
  // Do nothing if we're sharing a cuda managed pointer between control and execution:
  if (controlPtr == executionPtr && CudaAllocator::IsDevicePointer(controlPtr))
  {
    // If we're trying to copy a shared, non-managed device pointer back to
    // control throw an exception -- the pointer cannot be read from control,
    // so this operation is invalid.
    if (!CudaAllocator::IsManagedPointer(controlPtr))
    {
      throw vtkm::cont::ErrorBadValue(
        "Control pointer is a CUDA device pointer that does not supported managed access.");
    }

    // If it is managed, just return and let CUDA handle the migration for us.
    CudaAllocator::PrepareForControl(controlPtr, numBytes);
    return;
  }

  VTKM_CUDA_CALL(hipMemcpyAsync(controlPtr,
                                 executionPtr,
                                 static_cast<std::size_t>(numBytes),
                                 hipMemcpyDeviceToHost,
                                 hipStreamPerThread));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForRead(const void* controlPtr,
                                                                      const void* executionPtr,
                                                                      vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForInput(executionPtr, static_cast<size_t>(numBytes));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForWrite(const void* controlPtr,
                                                                       const void* executionPtr,
                                                                       vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForOutput(executionPtr, static_cast<size_t>(numBytes));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::UsingForReadWrite(
  const void* controlPtr,
  const void* executionPtr,
  vtkm::UInt64 numBytes) const
{
  CudaAllocator::PrepareForInPlace(executionPtr, static_cast<size_t>(numBytes));
}


} // end namespace internal

VTKM_INSTANTIATE_ARRAYHANDLES_FOR_DEVICE_ADAPTER(DeviceAdapterTagCuda)
}
} // end vtkm::cont
