//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 Sandia Corporation.
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#define vtk_m_cont_cuda_internal_ArrayManagerExecutionCuda_cu

#include <vtkm/cont/cuda/internal/ArrayManagerExecutionCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>

using vtkm::cont::cuda::internal::CudaAllocator;

namespace vtkm
{
namespace cont
{
namespace internal
{

ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::ExecutionArrayInterfaceBasic(
  StorageBasicBase& storage)
  : Superclass(storage)
{
}

DeviceAdapterId ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::GetDeviceId() const
{
  return VTKM_DEVICE_ADAPTER_CUDA;
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Allocate(TypelessExecutionArray& execArray,
                                                                  vtkm::Id numBytes) const
{
  if (execArray.Array != nullptr)
  {
    const vtkm::Id cap =
      static_cast<char*>(execArray.ArrayCapacity) - static_cast<char*>(execArray.Array);

    if (cap < numBytes)
    { // Current allocation too small -- free & realloc
      this->Free(execArray);
    }
    else
    { // Reuse buffer if possible:
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      return;
    }
  }

  VTKM_ASSERT(execArray.Array == nullptr);

  // Attempt to allocate:
  try
  {
    // Cast to char* so that the pointer math below will work.
    char* tmp = static_cast<char*>(CudaAllocator::Allocate(static_cast<size_t>(numBytes)));
    execArray.Array = tmp;
    execArray.ArrayEnd = tmp + numBytes;
    execArray.ArrayCapacity = tmp + numBytes;
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << numBytes << " bytes on device: " << error.what();
    throw vtkm::cont::ErrorBadAllocation(err.str());
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Free(
  TypelessExecutionArray& execArray) const
{
  if (execArray.Array != nullptr)
  {
    CudaAllocator::Free(execArray.Array);
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyFromControl(const void* controlPtr,
                                                                         void* executionPtr,
                                                                         vtkm::Id numBytes) const
{
  VTKM_CUDA_CALL(hipMemcpy(
    executionPtr, controlPtr, static_cast<std::size_t>(numBytes), hipMemcpyHostToDevice));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyToControl(const void* executionPtr,
                                                                       void* controlPtr,
                                                                       vtkm::Id numBytes) const
{
  VTKM_CUDA_CALL(hipMemcpy(
    controlPtr, executionPtr, static_cast<std::size_t>(numBytes), hipMemcpyDeviceToHost));
}

} // end namespace internal

VTKM_INSTANTIATE_ARRAYHANDLES_FOR_DEVICE_ADAPTER(DeviceAdapterTagCuda)
}
} // end vtkm::cont
