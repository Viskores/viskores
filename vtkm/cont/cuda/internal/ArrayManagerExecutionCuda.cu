//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 Sandia Corporation.
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#define vtk_m_cont_cuda_internal_ArrayManagerExecutionCuda_cu

#include <vtkm/cont/cuda/internal/ArrayManagerExecutionCuda.h>

namespace vtkm
{
namespace cont
{
namespace internal
{

ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::ExecutionArrayInterfaceBasic(
  StorageBasicBase& storage)
  : Superclass(storage)
{
}

DeviceAdapterId ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::GetDeviceId() const
{
  return VTKM_DEVICE_ADAPTER_CUDA;
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Allocate(TypelessExecutionArray& execArray,
                                                                  vtkm::Id numBytes) const
{
  if (execArray.Array != nullptr)
  {
    const vtkm::Id cap =
      static_cast<char*>(execArray.ArrayCapacity) - static_cast<char*>(execArray.Array);

    if (cap < numBytes)
    { // Current allocation too small -- free & realloc
      this->Free(execArray);
    }
    else
    { // Reuse buffer if possible:
      execArray.ArrayEnd = static_cast<char*>(execArray.Array) + numBytes;
      return;
    }
  }

  VTKM_ASSERT(execArray.Array == nullptr);

  // Attempt to allocate:
  try
  {
    char* tmp;
#ifdef VTKM_USE_UNIFIED_MEMORY
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    VTKM_CUDA_CALL(hipMallocManaged(&tmp, static_cast<std::size_t>(numBytes)));
    VTKM_CUDA_CALL(hipMemAdvise(tmp, numBytes, hipMemAdviseSetPreferredLocation, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(tmp, numBytes, dev, 0));
    VTKM_CUDA_CALL(hipStreamSynchronize(0));
#else
    VTKM_CUDA_CALL(hipMalloc(&tmp, static_cast<std::size_t>(numBytes)));
#endif

    execArray.Array = tmp;
    execArray.ArrayEnd = tmp + numBytes;
    execArray.ArrayCapacity = tmp + numBytes;
  }
  catch (const std::exception& error)
  {
    std::ostringstream err;
    err << "Failed to allocate " << numBytes << " bytes on device: " << error.what();
    throw vtkm::cont::ErrorBadAllocation(err.str());
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::Free(
  TypelessExecutionArray& execArray) const
{
  if (execArray.Array != nullptr)
  {
    VTKM_CUDA_CALL(hipFree(execArray.Array));
    execArray.Array = nullptr;
    execArray.ArrayEnd = nullptr;
    execArray.ArrayCapacity = nullptr;
  }
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyFromControl(const void* controlPtr,
                                                                         void* executionPtr,
                                                                         vtkm::Id numBytes) const
{
  VTKM_CUDA_CALL(hipMemcpy(
    executionPtr, controlPtr, static_cast<std::size_t>(numBytes), hipMemcpyHostToDevice));
}

void ExecutionArrayInterfaceBasic<DeviceAdapterTagCuda>::CopyToControl(const void* executionPtr,
                                                                       void* controlPtr,
                                                                       vtkm::Id numBytes) const
{
  VTKM_CUDA_CALL(hipMemcpy(
    controlPtr, executionPtr, static_cast<std::size_t>(numBytes), hipMemcpyDeviceToHost));
}

} // end namespace internal

VTKM_INSTANTIATE_ARRAYHANDLES_FOR_DEVICE_ADAPTER(DeviceAdapterTagCuda)
}
} // end vtkm::cont
