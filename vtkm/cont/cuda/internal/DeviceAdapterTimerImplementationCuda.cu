//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2014 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2014 UT-Battelle, LLC.
//  Copyright 2014 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================
#include <vtkm/cont/cuda/internal/DeviceAdapterTimerImplementationCuda.h>

#include <vtkm/Types.h>
#include <vtkm/cont/cuda/ErrorCuda.h>

#include <hip/hip_runtime.h>

namespace vtkm
{
namespace cont
{

DeviceAdapterTimerImplementation<
  vtkm::cont::DeviceAdapterTagCuda>::DeviceAdapterTimerImplementation()
{
  VTKM_CUDA_CALL(hipEventCreate(&this->StartEvent));
  VTKM_CUDA_CALL(hipEventCreate(&this->EndEvent));
  this->Reset();
}

DeviceAdapterTimerImplementation<
  vtkm::cont::DeviceAdapterTagCuda>::~DeviceAdapterTimerImplementation()
{
  // These aren't wrapped in VTKM_CUDA_CALL because we can't throw errors
  // from destructors. We're relying on hipGetLastError in the
  // VTKM_CUDA_CHECK_ASYNCHRONOUS_ERROR catching any issues from these calls
  // later.
  hipEventDestroy(this->StartEvent);
  hipEventDestroy(this->EndEvent);
}

void DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::Reset()
{
  VTKM_CUDA_CALL(hipEventRecord(this->StartEvent, hipStreamPerThread));
  VTKM_CUDA_CALL(hipEventSynchronize(this->StartEvent));
}

vtkm::Float64 DeviceAdapterTimerImplementation<vtkm::cont::DeviceAdapterTagCuda>::GetElapsedTime()
{
  VTKM_CUDA_CALL(hipEventRecord(this->EndEvent, hipStreamPerThread));
  VTKM_CUDA_CALL(hipEventSynchronize(this->EndEvent));
  float elapsedTimeMilliseconds;
  VTKM_CUDA_CALL(hipEventElapsedTime(&elapsedTimeMilliseconds, this->StartEvent, this->EndEvent));
  return static_cast<vtkm::Float64>(0.001f * elapsedTimeMilliseconds);
}
}
} // namespace vtkm::cont
