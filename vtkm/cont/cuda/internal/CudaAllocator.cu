//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//============================================================================

#include <cstdlib>
#include <mutex>
#include <vtkm/cont/Logging.h>
#include <vtkm/cont/RuntimeDeviceInformation.h>
#include <vtkm/cont/RuntimeDeviceTracker.h>
#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>
#include <vtkm/cont/cuda/internal/DeviceAdapterTagCuda.h>
#include <vtkm/cont/cuda/internal/RuntimeDeviceConfigurationCuda.h>
#define NO_VTKM_MANAGED_MEMORY "NO_VTKM_MANAGED_MEMORY"

#include <cstdlib>
#include <mutex>
#include <vector>

VTKM_THIRDPARTY_PRE_INCLUDE
#include <hip/hip_runtime.h>
VTKM_THIRDPARTY_POST_INCLUDE

// These static vars are in an anon namespace to work around MSVC linker issues.
namespace
{
// Has CudaAllocator::Initialize been called by any thread?
static std::once_flag IsInitializedFlag;

// Used to keep track of whether the CUDA allocator has been initialized CUDA has not
// been finalized (since CUDA does not seem to track that for us).
static bool IsInitialized = false;

// Holds how VTK-m currently allocates memory.
// When VTK-m is initialized we set this based on the hardware support ( HardwareSupportsManagedMemory ).
// The user can explicitly disable managed memory through an enviornment variable
// or by calling a function on the CudaAllocator.
// Likewise managed memory can be re-enabled by calling a function on CudaAllocator
// if and only if the underlying hardware supports pageable managed memory
static bool ManagedMemoryEnabled = false;

// True if concurrent pagable managed memory is supported by the machines hardware.
static bool HardwareSupportsManagedMemory = false;

// Avoid overhead of hipMemAdvise and hipMemPrefetchAsync for small buffers.
// This value should be > 0 or else these functions will error out.
static std::size_t Threshold = 1 << 20;
}

namespace vtkm
{
namespace cont
{
namespace cuda
{
namespace internal
{

bool CudaAllocator::UsingManagedMemory()
{
  CudaAllocator::Initialize();
  return ManagedMemoryEnabled;
}

void CudaAllocator::ForceManagedMemoryOff()
{
  if (HardwareSupportsManagedMemory)
  {
    ManagedMemoryEnabled = false;
    VTKM_LOG_F(vtkm::cont::LogLevel::Info, "CudaAllocator disabling managed memory");
  }
  else
  {
    VTKM_LOG_F(
      vtkm::cont::LogLevel::Warn,
      "CudaAllocator trying to disable managed memory on hardware that doesn't support it");
  }
}

void CudaAllocator::ForceManagedMemoryOn()
{
  if (HardwareSupportsManagedMemory)
  {
    ManagedMemoryEnabled = true;
    VTKM_LOG_F(vtkm::cont::LogLevel::Info, "CudaAllocator enabling managed memory");
  }
  else
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::Warn,
               "CudaAllocator trying to enable managed memory on hardware that doesn't support it");
  }
}

bool CudaAllocator::IsDevicePointer(const void* ptr)
{
  CudaAllocator::Initialize();
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.devicePointer == ptr;
}

bool CudaAllocator::IsManagedPointer(const void* ptr)
{
  if (!ptr || !ManagedMemoryEnabled)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
#if CUDART_VERSION < 10000 // isManaged deprecated in CUDA 10.
  return attr.isManaged != 0;
#else // attr.type doesn't exist before CUDA 10
  return attr.type == hipMemoryTypeManaged;
#endif
}

void* CudaAllocator::Allocate(std::size_t numBytes)
{
  CudaAllocator::Initialize();
  // When numBytes is zero hipMallocManaged returns an error and the behavior
  // of hipMalloc is not documented. Just return nullptr.
  if (numBytes == 0)
  {
    return nullptr;
  }

  void* ptr = nullptr;
#if CUDART_VERSION >= 11030
  const auto& tracker = vtkm::cont::GetRuntimeDeviceTracker();
  if (tracker.GetThreadFriendlyMemAlloc())
  {
    VTKM_CUDA_CALL(hipMallocAsync(&ptr, numBytes, hipStreamPerThread));
  }
  else
#endif
    if (ManagedMemoryEnabled)
  {
    VTKM_CUDA_CALL(hipMallocManaged(&ptr, numBytes));
  }
  else
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Allocated CUDA array of %s at %p.",
               vtkm::cont::GetSizeString(numBytes).c_str(),
               ptr);
  }

  return ptr;
}

void* CudaAllocator::AllocateUnManaged(std::size_t numBytes)
{
  void* ptr = nullptr;
#if CUDART_VERSION >= 11030
  const auto& tracker = vtkm::cont::GetRuntimeDeviceTracker();
  if (tracker.GetThreadFriendlyMemAlloc())
  {
    VTKM_CUDA_CALL(hipMallocAsync(&ptr, numBytes, hipStreamPerThread));
  }
  else
#endif
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Allocated CUDA array of %s at %p.",
               vtkm::cont::GetSizeString(numBytes).c_str(),
               ptr);
  }
  return ptr;
}

void CudaAllocator::Free(void* ptr)
{
  if (!IsInitialized)
  {
    // Since the data was successfully allocated, it is a fair assumption that the CUDA
    // runtime has been finalized and a global object is trying to destroy itself. Since
    // CUDA already cleaned up all memory for program exit, we can ignore this free.
    return;
  }

  VTKM_LOG_F(vtkm::cont::LogLevel::MemExec, "Freeing CUDA allocation at %p.", ptr);

#if CUDART_VERSION >= 11030
  const auto& tracker = vtkm::cont::GetRuntimeDeviceTracker();
  if (tracker.GetThreadFriendlyMemAlloc())
  {
    VTKM_CUDA_CALL(hipFreeAsync(ptr, hipStreamPerThread));
  }
  else
#endif
  {
    VTKM_CUDA_CALL(hipFree(ptr));
  }
}

void CudaAllocator::FreeDeferred(void* ptr, std::size_t numBytes)
{
  if (!IsInitialized)
  {
    // Since the data was successfully allocated, it is a fair assumption that the CUDA
    // runtime has been finalized and a global object is trying to destroy itself. Since
    // CUDA already cleaned up all memory for program exit, we can ignore this free.
    return;
  }

  static std::mutex deferredMutex;
  static std::vector<void*> deferredPointers;
  static std::size_t deferredSize = 0;
  constexpr std::size_t bufferLimit = 2 << 24; //16MB buffer

  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec,
               "Deferring free of CUDA allocation at %p of %s.",
               ptr,
               vtkm::cont::GetSizeString(numBytes).c_str());
  }

  std::vector<void*> toFree;
  // critical section
  {
    std::lock_guard<std::mutex> lock(deferredMutex);
    deferredPointers.push_back(ptr);
    deferredSize += numBytes;
    if (deferredSize >= bufferLimit)
    {
      toFree.swap(deferredPointers);
      deferredSize = 0;
    }
  }

  for (auto&& p : toFree)
  {
    VTKM_LOG_F(vtkm::cont::LogLevel::MemExec, "Freeing deferred CUDA allocation at %p.", p);
    VTKM_CUDA_CALL(hipFree(p));
  }
}

void CudaAllocator::PrepareForControl(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
    // TODO these hints need to be benchmarked and adjusted once we start
    // sharing the pointers between cont/exec
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, hipCpuDeviceId, hipStreamPerThread));
  }
}

void CudaAllocator::PrepareForInput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
    vtkm::Id dev;
    vtkm::cont::RuntimeDeviceInformation()
      .GetRuntimeConfiguration(vtkm::cont::DeviceAdapterTagCuda())
      .GetDeviceInstance(dev);
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
  }
}

void CudaAllocator::PrepareForOutput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
    vtkm::Id dev;
    vtkm::cont::RuntimeDeviceInformation()
      .GetRuntimeConfiguration(vtkm::cont::DeviceAdapterTagCuda())
      .GetDeviceInstance(dev);
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
  }
}

void CudaAllocator::PrepareForInPlace(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
    vtkm::Id dev;
    vtkm::cont::RuntimeDeviceInformation()
      .GetRuntimeConfiguration(vtkm::cont::DeviceAdapterTagCuda())
      .GetDeviceInstance(dev);
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
  }
}

void CudaAllocator::Initialize()
{
  std::call_once(IsInitializedFlag, []() {
    auto cudaDeviceConfig = dynamic_cast<
      vtkm::cont::internal::RuntimeDeviceConfiguration<vtkm::cont::DeviceAdapterTagCuda>&>(
      vtkm::cont::RuntimeDeviceInformation{}.GetRuntimeConfiguration(
        vtkm::cont::DeviceAdapterTagCuda()));
    vtkm::Id numDevices;
    cudaDeviceConfig.GetMaxDevices(numDevices);

    if (numDevices == 0)
    {
      return;
    }

    // Check all devices, use the feature set supported by all
    bool managedMemorySupported = true;
    std::vector<hipDeviceProp_t> cudaProp;
    cudaDeviceConfig.GetCudaDeviceProp(cudaProp);
    for (int i = 0; i < numDevices && managedMemorySupported; ++i)
    {
      // We check for concurrentManagedAccess, as devices with only the
      // managedAccess property have extra synchronization requirements.
      managedMemorySupported = managedMemorySupported && cudaProp[i].concurrentManagedAccess;
    }

    HardwareSupportsManagedMemory = managedMemorySupported;
    ManagedMemoryEnabled = managedMemorySupported;

    VTKM_LOG_F(vtkm::cont::LogLevel::Info,
               "CudaAllocator hardware %s managed memory",
               HardwareSupportsManagedMemory ? "supports" : "doesn't support");

// Check if users want to disable managed memory
#pragma warning(push)
// getenv is not thread safe on windows but since it's inside a call_once block so
// it's fine to suppress the warning here.
#pragma warning(disable : 4996)
    const char* buf = std::getenv(NO_VTKM_MANAGED_MEMORY);
#pragma warning(pop)
    if (managedMemorySupported && buf != nullptr)
    { //only makes sense to disable managed memory if the hardware supports it
      //in the first place
      ManagedMemoryEnabled = false;
      VTKM_LOG_F(
        vtkm::cont::LogLevel::Info,
        "CudaAllocator disabling managed memory due to NO_VTKM_MANAGED_MEMORY env variable");
    }

    // CUDA does not give any indication of whether it is still running, but we have found from
    // experience that it finalizes itself during program termination. However, the user might
    // have their own objects being cleaned up during termination after CUDA. We need a flag
    // to catch if this happens after CUDA finalizes itself. We will set this flag to true now
    // and false on termination. Because we are creating the atexit call here (after CUDA must
    // have initialized itself), C++ will require our function that unsets the flag to happen
    // before CUDA finalizes.
    IsInitialized = true;
    std::atexit([]() { IsInitialized = false; });
  });
}
}
}
}
} // end namespace vtkm::cont::cuda::internal
