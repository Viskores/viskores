//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>

VTKM_THIRDPARTY_PRE_INCLUDE
#include <hip/hip_runtime.h>
VTKM_THIRDPARTY_POST_INCLUDE

// These static vars are in an anon namespace to work around MSVC linker issues.
namespace
{
// Has CudaAllocator::Initialize been called?
static bool IsInitialized = false;

// True if all devices support concurrent pagable managed memory.
static bool ManagedMemorySupported = false;
}

namespace vtkm
{
namespace cont
{
namespace cuda
{
namespace internal
{

bool CudaAllocator::UsingManagedMemory()
{
  CudaAllocator::Initialize();
  return ManagedMemorySupported;
}

bool CudaAllocator::IsDevicePointer(const void* ptr)
{
  CudaAllocator::Initialize();
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.devicePointer == ptr;
}

bool CudaAllocator::IsManagedPointer(const void* ptr)
{
  if (!ptr || !ManagedMemorySupported)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.isManaged != 0;
}

void* CudaAllocator::Allocate(std::size_t numBytes)
{
  CudaAllocator::Initialize();

  void* ptr = nullptr;
  if (ManagedMemorySupported)
  {
    VTKM_CUDA_CALL(hipMallocManaged(&ptr, numBytes));
  }
  else
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  return ptr;
}

void CudaAllocator::Free(void* ptr)
{
  VTKM_CUDA_CALL(hipFree(ptr));
}

void CudaAllocator::PrepareForControl(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr))
  {
#if CUDART_VERSION >= 8000
    // TODO these hints need to be benchmarked and adjusted once we start
    // sharing the pointers between cont/exec
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, hipCpuDeviceId, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr))
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForOutput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr))
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInPlace(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr))
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::Initialize()
{
  if (!IsInitialized)
  {
    int numDevices;
    VTKM_CUDA_CALL(hipGetDeviceCount(&numDevices));

    if (numDevices == 0)
    {
      ManagedMemorySupported = false;
      IsInitialized = true;
      return;
    }

    // Check all devices, use the feature set supported by all
    bool managed = true;
    hipDeviceProp_t prop;
    for (int i = 0; i < numDevices && managed; ++i)
    {
      VTKM_CUDA_CALL(hipGetDeviceProperties(&prop, i));
      // We check for concurrentManagedAccess, as devices with only the
      // managedAccess property have extra synchronization requirements.
      managed = managed && prop.concurrentManagedAccess;
    }

    ManagedMemorySupported = managed;
    IsInitialized = true;
  }
}
}
}
}
} // end namespace vtkm::cont::cuda::internal
