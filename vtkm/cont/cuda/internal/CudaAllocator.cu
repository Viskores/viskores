//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>

VTKM_THIRDPARTY_PRE_INCLUDE
#include <hip/hip_runtime.h>
VTKM_THIRDPARTY_POST_INCLUDE

// These static vars are in an anon namespace to work around MSVC linker issues.
namespace
{
#if CUDART_VERSION >= 8000
// Has CudaAllocator::Initialize been called?
static bool IsInitialized = false;
#endif

// True if all devices support concurrent pagable managed memory.
static bool ManagedMemorySupported = false;

// Avoid overhead of hipMemAdvise and hipMemPrefetchAsync for small buffers.
// This value should be > 0 or else these functions will error out.
static std::size_t Threshold = 1 << 20;
}

namespace vtkm
{
namespace cont
{
namespace cuda
{
namespace internal
{

bool CudaAllocator::UsingManagedMemory()
{
  CudaAllocator::Initialize();
  return ManagedMemorySupported;
}

bool CudaAllocator::IsDevicePointer(const void* ptr)
{
  CudaAllocator::Initialize();
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.devicePointer == ptr;
}

bool CudaAllocator::IsManagedPointer(const void* ptr)
{
  if (!ptr || !ManagedMemorySupported)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.isManaged != 0;
}

void* CudaAllocator::Allocate(std::size_t numBytes)
{
  CudaAllocator::Initialize();
  // When numBytes is zero hipMallocManaged returns an error and the behavior
  // of hipMalloc is not documented. Just return nullptr.
  if (numBytes == 0)
  {
    return nullptr;
  }

  void* ptr = nullptr;
  if (ManagedMemorySupported)
  {
    VTKM_CUDA_CALL(hipMallocManaged(&ptr, numBytes));
  }
  else
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  return ptr;
}

void CudaAllocator::Free(void* ptr)
{
  VTKM_CUDA_CALL(hipFree(ptr));
}

void CudaAllocator::PrepareForControl(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    // TODO these hints need to be benchmarked and adjusted once we start
    // sharing the pointers between cont/exec
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, hipCpuDeviceId, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForOutput(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::PrepareForInPlace(const void* ptr, std::size_t numBytes)
{
  if (IsManagedPointer(ptr) && numBytes >= Threshold)
  {
#if CUDART_VERSION >= 8000
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    // VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetAccessedBy, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, hipStreamPerThread));
#endif // CUDA >= 8.0
  }
}

void CudaAllocator::Initialize()
{
#if CUDART_VERSION >= 8000
  if (!IsInitialized)
  {
    int numDevices;
    VTKM_CUDA_CALL(hipGetDeviceCount(&numDevices));

    if (numDevices == 0)
    {
      ManagedMemorySupported = false;
      IsInitialized = true;
      return;
    }

    // Check all devices, use the feature set supported by all
    bool managed = true;
    hipDeviceProp_t prop;
    for (int i = 0; i < numDevices && managed; ++i)
    {
      VTKM_CUDA_CALL(hipGetDeviceProperties(&prop, i));
      // We check for concurrentManagedAccess, as devices with only the
      // managedAccess property have extra synchronization requirements.
      managed = managed && prop.concurrentManagedAccess;
    }

    ManagedMemorySupported = managed;
    IsInitialized = true;
  }
#else
  ManagedMemorySupported = false;
#endif
}
}
}
}
} // end namespace vtkm::cont::cuda::internal
