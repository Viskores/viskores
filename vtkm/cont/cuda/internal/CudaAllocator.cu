//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2017 Sandia Corporation.
//  Copyright 2017 UT-Battelle, LLC.
//  Copyright 2017 Los Alamos National Security.
//
//  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#include <vtkm/cont/cuda/ErrorCuda.h>
#include <vtkm/cont/cuda/internal/CudaAllocator.h>

#include <hip/hip_runtime.h>

// These static vars are in an anon namespace to work around MSVC linker issues.
namespace
{
// Has CudaAllocator::Initialize been called?
static bool IsInitialized = false;

// True if all devices support concurrent pagable managed memory.
static bool ManagedMemorySupported = false;
}

namespace vtkm
{
namespace cont
{
namespace cuda
{
namespace internal
{

bool CudaAllocator::UsingManagedMemory()
{
  CudaAllocator::Initialize();
  return ManagedMemorySupported;
}

bool CudaAllocator::IsDevicePointer(const void* ptr)
{
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.devicePointer == ptr;
}

bool CudaAllocator::IsManagedPointer(const void* ptr)
{
  if (!ptr)
  {
    return false;
  }

  hipPointerAttribute_t attr;
  hipError_t err = hipPointerGetAttributes(&attr, ptr);
  // This function will return invalid value if the pointer is unknown to the
  // cuda runtime. Manually catch this value since it's not really an error.
  if (err == hipErrorInvalidValue)
  {
    hipGetLastError(); // Clear the error so we don't raise it later...
    return false;
  }
  VTKM_CUDA_CALL(err /*= hipPointerGetAttributes(&attr, ptr)*/);
  return attr.isManaged != 0;
}

void* CudaAllocator::Allocate(std::size_t numBytes)
{
  CudaAllocator::Initialize();

  void* ptr = nullptr;
  if (ManagedMemorySupported)
  {
    VTKM_CUDA_CALL(hipMallocManaged(&ptr, numBytes));
  }
  else
  {
    VTKM_CUDA_CALL(hipMalloc(&ptr, numBytes));
  }

  return ptr;
}

void CudaAllocator::Free(void* ptr)
{
  CudaAllocator::Initialize();

  VTKM_CUDA_CALL(hipFree(ptr));
}

void CudaAllocator::PrepareForControl(const void* ptr, std::size_t numBytes)
{
  CudaAllocator::Initialize();

  if (ManagedMemorySupported)
  {
    // TODO these hints need to be benchmarked and adjusted once we start
    // sharing the pointers between cont/exec
    VTKM_CUDA_CALL(
      hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, hipCpuDeviceId));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, hipCpuDeviceId, 0));
  }
}

void CudaAllocator::PrepareForInput(const void* ptr, std::size_t numBytes)
{
  CudaAllocator::Initialize();

  if (ManagedMemorySupported)
  {
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, 0));
  }
}

void CudaAllocator::PrepareForOutput(const void* ptr, std::size_t numBytes)
{
  CudaAllocator::Initialize();

  if (ManagedMemorySupported)
  {
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, 0));
  }
}

void CudaAllocator::PrepareForInPlace(const void* ptr, std::size_t numBytes)
{
  CudaAllocator::Initialize();

  if (ManagedMemorySupported)
  {
    int dev;
    VTKM_CUDA_CALL(hipGetDevice(&dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseSetPreferredLocation, dev));
    VTKM_CUDA_CALL(hipMemAdvise(ptr, numBytes, hipMemAdviseUnsetReadMostly, dev));
    VTKM_CUDA_CALL(hipMemPrefetchAsync(ptr, numBytes, dev, 0));
  }
}

void CudaAllocator::Initialize()
{
  if (!IsInitialized)
  {
    int numDevices;
    VTKM_CUDA_CALL(hipGetDeviceCount(&numDevices));

    if (numDevices == 0)
    {
      ManagedMemorySupported = false;
      IsInitialized = true;
      return;
    }

    // Check all devices, use the feature set supported by all
    bool managed = true;
    hipDeviceProp_t prop;
    for (int i = 0; i < numDevices && managed; ++i)
    {
      VTKM_CUDA_CALL(hipGetDeviceProperties(&prop, i));
      // We check for concurrentManagedAccess, as devices with only the
      // managedAccess property have extra synchronization requirements.
      managed = managed && prop.concurrentManagedAccess;
    }

    ManagedMemorySupported = managed;
    IsInitialized = true;
  }
}
}
}
}
} // end namespace vtkm::cont::cuda::internal
