#include "hip/hip_runtime.h"
//=============================================================================
//
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2015 National Technology & Engineering Solutions of Sandia, LLC (NTESS).
//  Copyright 2015 UT-Battelle, LLC.
//  Copyright 2015 Los Alamos National Security.
//
//  Under the terms of Contract DE-NA0003525 with NTESS,
//  the U.S. Government retains certain rights in this software.
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//
//=============================================================================

// Make sure that the tested code is using the device adapter specified. This
// is important in the long run so we don't, for example, use the CUDA device
// for a part of an operation where the TBB device was specified.
#define VTKM_DEVICE_ADAPTER VTKM_DEVICE_ADAPTER_ERROR

#include <vtkm/cont/RuntimeDeviceTracker.h>
#include <vtkm/cont/cuda/DeviceAdapterCuda.h>
#include <vtkm/testing/TestingMath.h>

#include <vtkm/worklet/DispatcherMapField.h>
#include <vtkm/worklet/WorkletMapField.h>

#include <cmath>
#include <iostream>
#include <limits>
#include <random>
#include <vector>

#include "hiprand/hiprand_kernel.h"

namespace
{

struct TriggerICE : public vtkm::worklet::WorkletMapField
{
  using ControlSignature = void(FieldIn<>, FieldIn<>, FieldOut<>);
  using ExecutionSignature = _3(_1, _2, WorkIndex);

#if __CUDA_ARCH__
  template <class ValueType>
  __device__ ValueType operator()(const ValueType& bad,
                                  const ValueType& sane,
                                  const vtkm::Id sequenceId) const
  {

    hiprandState_t state;
    //Each thread uses same seed but different sequence numbers
    hiprand_init(42, sequenceId, 0, &state);

    int signBad = vtkm::SignBit(bad);
    int signGood = vtkm::SignBit(bad);

    vtkm::Vec<ValueType, 3> coord = { vtkm::Abs(bad * sane),
                                      bad * sane + (ValueType)signBad,
                                      bad * sane + (ValueType)signGood };

    for (int i = 0; i < 10; ++i)
    {
      for (int j = 0; j < 3; ++j)
      {
        if (vtkm::IsNan(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 5.0f;
          coord[j] = vtkm::Sqrt(vtkm::Dot(coord, coord));
          if (coord[j] <= 1.0f)
          {
            coord[j] += 1.0f;
          }
        }
        if (vtkm::IsInf(coord[j]))
        {
          coord[j] = hiprand_normal(&state) * 8.0f;
          coord[j] = vtkm::Tan(vtkm::Cos(vtkm::Dot(coord, coord)));
        }
      }
    }
    return coord[0] * 4.0f + coord[1] * 4.0f + coord[2] * 4.0f;
  }
#else
  template <class ValueType>
  ValueType operator()(const ValueType& bad, const ValueType& sane, const vtkm::Id sequenceId) const
  {
    return bad + sane * static_cast<ValueType>(sequenceId);
  }
#endif
};

//-----------------------------------------------------------------------------
template <typename Device>
void RunEdgeCases()
{
  std::cout << "Testing complicated worklets that can cause NVCC to ICE." << std::endl;
  //When running CUDA on unsupported hardware we find that IsInf, IsNan, and
  //SignBit can cause the CUDA compiler to crash. This test is a consistent
  //way to detect this.
  //
  //The way it works is we generate all kinds of nasty floating point values
  //such as signaling Nan, quiet Nan, other Nans, +Inf, -Inf, -0, +0, a collection of
  //denormal numbers, and the min and max float values
  //and than a random collection of values from normal float space. We combine this
  //array which we will call 'bad' with another input array which we will call 'sane',
  //We than execute a worklet that takes values stored in 'bad' and 'sane' that does
  //some computation that takes into account the results of IsInf, IsNan, and
  //SignBit
  const vtkm::Id desired_size = 2048;
  std::vector<float> sanevalues;
  std::vector<float> badvalues = { std::numeric_limits<float>::signaling_NaN(),
                                   std::numeric_limits<float>::quiet_NaN(),
                                   std::nanf("1"),
                                   std::nanf("4200042"),
                                   std::numeric_limits<float>::infinity(),
                                   std::numeric_limits<float>::infinity() * -1,
                                   0.0f,
                                   -0.0f,
                                   std::numeric_limits<float>::denorm_min(),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f),
                                   std::numeric_limits<float>::denorm_min() *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::nextafter(std::numeric_limits<float>::min(), 0.0f) *
                                     (1 + std::numeric_limits<float>::epsilon()),
                                   std::numeric_limits<float>::lowest(),
                                   std::numeric_limits<float>::min(),
                                   std::numeric_limits<float>::max() };
  const std::size_t bad_size = badvalues.size();
  const vtkm::Id bad_size_as_id = static_cast<vtkm::Id>(bad_size);

  badvalues.reserve(desired_size);
  sanevalues.reserve(desired_size);

  //construct a random number generator
  std::mt19937 rng;
  std::uniform_real_distribution<float> range(-1.0f, 1.0f);

  // now add in some random numbers to the bad values
  for (std::size_t i = 0; i < desired_size - bad_size; ++i)
  {
    badvalues.push_back(range(rng));
  }
  for (std::size_t i = 0; i < desired_size; ++i)
  {
    sanevalues.push_back(range(rng));
  }

  auto bad = vtkm::cont::make_ArrayHandle(badvalues);
  auto sane = vtkm::cont::make_ArrayHandle(sanevalues);
  decltype(sane) result;
  vtkm::worklet::DispatcherMapField<TriggerICE> dispatcher;
  dispatcher.SetDevice(Device());
  dispatcher.Invoke(bad, sane, result);

  auto portal = result.GetPortalConstControl();

  //the first 6 values should be nan
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(0)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(1)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(2)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(3)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(4)), "Value should be NaN.");
  VTKM_TEST_ASSERT(vtkm::IsNan(portal.Get(5)), "Value should be NaN.");

  for (vtkm::Id i = bad_size_as_id; i < desired_size; ++i)
  { //The rest of the values shouldn't be Nan or Inf
    auto v = portal.Get(i);
    const bool valid = !vtkm::IsNan(v) && !vtkm::IsInf(v);
    VTKM_TEST_ASSERT(valid, "value shouldn't be NaN or INF");
  }
}

} //namespace

int UnitTestCudaMath(int, char* [])
{
  auto tracker = vtkm::cont::GetGlobalRuntimeDeviceTracker();
  tracker.ForceDevice(vtkm::cont::DeviceAdapterTagCuda{});
  int tests_valid = vtkm::cont::testing::Testing::Run(
    UnitTestMathNamespace::RunMathTests<vtkm::cont::DeviceAdapterTagCuda>);

  tests_valid += vtkm::cont::testing::Testing::Run(RunEdgeCases<vtkm::cont::DeviceAdapterTagCuda>);

  return tests_valid;
}
